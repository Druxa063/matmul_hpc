#include <stdio.h>
#include <assert.h>
#include <iostream>
#include <chrono>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>


template <int BLOCK_SIZE> __global__ void MatrixMulCUDA(float *C, float *A, float *B, int wA, int wB) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int aBegin = wA * BLOCK_SIZE * by;
    int aEnd   = aBegin + wA - 1;
    int aStep  = BLOCK_SIZE;

    int bBegin = BLOCK_SIZE * bx;
    int bStep  = BLOCK_SIZE * wB;

    float Csub = 0;

    for (int a = aBegin, b = bBegin;
            a <= aEnd;
            a += aStep, b += bStep) {
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];


        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

    // Synchronize to make sure the matrices are loaded
        __syncthreads();
#pragma unroll

        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub += As[ty][k] * Bs[k][tx];
        }

        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}


void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
     // Create a pseudo-random number generator
     hiprandGenerator_t prng;
     hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

     // Set the seed for the random number generator using the system clock
     hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

     // Fill the array with random numbers on the device
     hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
 }

int MatrixMultiply(int argc, char **argv,
                   int block_size, const dim3 &dimsA,
                   const dim3 &dimsB) {
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = reinterpret_cast<float *>(malloc(mem_size_A));
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = reinterpret_cast<float *>(malloc(mem_size_B));

    // Allocate device memory
    float *d_A, *d_B, *d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float *h_C = reinterpret_cast<float *>(malloc(mem_size_C));
    float *h_C_test = reinterpret_cast<float *>(malloc(mem_size_C));

    for (int i = 0; i < static_cast<int>(dimsC.x * dimsC.y); i++) {
        h_C_test[i]=0;
    }

    if (h_C == NULL) {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    hipMalloc(&d_A, mem_size_A);

    hipMalloc(&d_B, mem_size_B);

    hipMalloc(&d_C, mem_size_C);


    GPU_fill_rand(d_A, dimsA.x, dimsA.y);
    GPU_fill_rand(d_B, dimsB.x, dimsB.y);

    // Only for tests purposes
    hipMemcpy(h_A,d_A, mem_size_A,hipMemcpyDeviceToHost);
    hipMemcpy(h_B,d_B, mem_size_A,hipMemcpyDeviceToHost);

    // Setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

    // Create and start timer
    printf("Computing result using CUDA Kernel...\n");

    hipDeviceSynchronize();

    // Allocate CUDA events that we'll use for timing
    hipEvent_t start;
    hipEventCreate(&start);

    hipEvent_t stop;
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, NULL);


    if (block_size == 16) {
        MatrixMulCUDA<16> <<< grid, threads >>>(d_C, d_A, d_B,
                                                    dimsA.x, dimsB.x);
    } else {
        MatrixMulCUDA<32> <<< grid, threads >>>(d_C, d_A, d_B,
                                                    dimsA.x, dimsB.x);
    }


    // Record the stop event
    hipEventRecord(stop, NULL);

    // Wait for the stop event to complete
    hipEventSynchronize(stop);

    printf("done\n");

    float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);

    // Compute and print the performance
    float msecPerMatrixMul = msecTotal;


    printf(
        "Time= %.3f msec," \
        " WorkgroupSize= %u threads/block\n",
        msecPerMatrixMul,
        threads.x * threads.y);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

    bool correct = true;


    auto t1 = std::chrono::high_resolution_clock::now();
    for(int i = 0; i < dimsA.x; ++i)
        for(int j = 0; j < dimsA.y; ++j)
            for(int k = 0; k < dimsA.x; ++k)
            {
                h_C_test[j + i * dimsA.x] += h_A[i * dimsA.x + k] * h_B[k* dimsA.x +j];
            }

    auto t2 = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(t2-t1).count();
    std::cout <<  duration;

    printf("Checking computed result for correctness: ");

    // test relative error by the formula
    //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
    double eps = 1.e-6;  // machine zero
    const float valB = 0.01f;
    for (int i = 0; i < static_cast<int>(dimsC.x * dimsC.y); i++) {
        double abs_err = fabs(h_C[i] - h_C_test[i]);
        double dot_length = dimsA.x;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err / abs_val / dot_length;

        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                   i, h_C[i], dimsA.x * valB, eps);
            correct = false;
            break;
        }
    }

    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    if (correct) {
        return EXIT_SUCCESS;
    } else {
        return EXIT_FAILURE;
    }
}

int main(int argc, char **argv) {
    printf("[Matrix Multiply Using CUDA] - Starting...\n");

    int block_size = 32;

    dim3 dimsA(50 * block_size, 50 * block_size, 1);
    dim3 dimsB(50 * block_size, 50 * block_size, 1);


    printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y,
                                               dimsB.x, dimsB.y);

    int matrix_result = MatrixMultiply(argc, argv, block_size, dimsA, dimsB);

    exit(matrix_result);
}


